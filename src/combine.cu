#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define BLOCK_DIM 1024
#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }

}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix. Matrix a and b are both in a batch
   * format, with shape [batch_size, m, n], [batch_size, n, p].
   * Requirements:
   * - All data must be first moved to shared memory.
   * - Only read each cell in a and b once.
   * - Only write to global memory once per kernel.
   * There is guarantee that a_shape[0] == b_shape[0], a_shape[2] == b_shape[1],
   * and out_shape[0] == a_shape[0], out_shape[1] == b_shape[1]
   *
   * Args:
   *   out: compact 1D array of size batch_size x m x p to write the output to
   *   out_shape: shape of the output array
   *   out_strides: strides of the output array
   *   a_storage: compact 1D array of size batch_size x m x n
   *   a_shape: shape of the a array
   *   a_strides: strides of the a array
   *   b_storage: comapct 2D array of size batch_size x n x p
   *   b_shape: shape of the b array
   *   b_strides: strides of the b array
   *
   * Returns:
   *   None (Fills in out array)
   */

    __shared__ float a_shared[TILE][TILE];
    __shared__ float b_shared[TILE][TILE];

    // In each block, we will compute a batch of the output matrix
    // All the threads in the block will work together to compute this batch
    int batch = blockIdx.z;
    int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
    int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;

    // 1. Compute the row and column of the output matrix this block will compute
    int out_row = threadIdx.x + blockIdx.x * blockDim.x;
    int out_col = threadIdx.y + blockIdx.y * blockDim.y;
    if ((batch >= out_shape[0]) || (out_row >= out_shape[1]) || (out_col >= out_shape[2])) return;

    // 2. Compute the position in the output array that this thread will write to
    int out_position = batch * out_strides[0] + out_row * out_strides[1] + out_col * out_strides[2];

    // 3. Iterate over tiles of the two input matrices, read the data into shared memory
    int tile_i = blockIdx.x * blockDim.x;
    int tile_k = blockIdx.y * blockDim.y;
    int thread_i = threadIdx.x;
    int thread_k = threadIdx.y;
    float out_ik = 0;
    __syncthreads();
    for (int tile_j = 0; tile_j < a_shape[2]; tile_j += TILE) {
      for (int j = 0; j < TILE; j++) {
        if (tile_j + j >= a_shape[2]) break;
        a_shared[thread_i][j] = a_storage[batch * a_batch_stride + (tile_i + thread_i) * a_strides[1] + (tile_j + j) * a_strides[2]];
        b_shared[j][thread_k] = b_storage[batch * b_batch_stride + (tile_j + j) * b_strides[1] + (tile_k + thread_k) * b_strides[2]];
      }

      // 4. Synchronize to make sure the data is available to all threads
      __syncthreads();

      // 5. Compute the output tile for this thread block
      for (int j = 0; j < TILE; j++) {
        if (tile_j + j >= a_shape[2]) break;
        out_ik += a_shared[thread_i][j] * b_shared[j][thread_k];
      }

      // 6. Synchronize to make sure all threads are done computing the output tile for (row, col)
      __syncthreads();
    }

    // 7. Write the output to global memory
    out[out_position] = out_ik;
}


__global__ void mapKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* in_storage,
    int* in_shape,
    int* in_strides,
    int shape_size,
    int fn_id
) {
  /**
   * Map function. Apply a unary function to each element of the input array and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  in_storage: compact 1D array of size in_size
   *  in_shape: shape of the input array
   *  in_strides: strides of the input array
   *  shape_size: number of dimensions in the input and output arrays, assume dimensions are the same
   *  fn_id: id of the function to apply to each element of the input array
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];

    // 1. Compute the position in the output array that this thread will write to
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= out_size) return;

    // 2. Convert the position to the out_index according to out_shape
    to_index(idx, out_shape, out_index, shape_size);
    for (int i = 0; i < shape_size; i++) {
        if (out_index[i] >= out_shape[i]) {
          return;
	    }
    }

    // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
    broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
    for (int i = 0; i < shape_size; i++) {
        if (in_index[i] >= in_shape[i]) {
	        return;
	      }
    }

    // 4. Calculate the position of element in in_array according to in_index and in_strides
    int in_position = index_to_position(in_index, in_strides, shape_size);

    // 5. Calculate the position of element in out_array according to out_index and out_strides
    int out_position = index_to_position(out_index, out_strides, shape_size);

    // 6. Apply the unary function to the input element and write the output to the out memory
    out[out_position] = fn(fn_id, in_storage[in_position]);
}

__global__ void reduceKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
  /**
   * Reduce function. Apply a reduce function to elements of the input array a and store the result in the output array.
   * Optimization:
   * Parallelize over the reduction operation. Each kernel performs one reduction.
   * e.g. a = [[1, 2, 3], [4, 5, 6]], kernel0 computes reduce([1, 2, 3]), kernel1 computes reduce([4, 5, 6]).
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  reduce_dim: dimension to reduce on
   *  reduce_value: initial value for the reduction
   *  shape_size: number of dimensions in the input & output array, assert dimensions are the same
   *  fn_id: id of the reduce function, currently only support add, multiply, and max
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];

    // 1. Define the position of the output element that this thread or this block will write to
    int idx = blockIdx.x;
    if (idx >= out_size || threadIdx.x >= a_shape[reduce_dim]) return;

    // 2. Convert the out_pos to the out_index according to out_shape
    to_index(idx, out_shape, out_index, shape_size);
    for (int i = 0; i < shape_size; i++) {
      if (out_index[i] >= out_shape[i]) {
        return;
      }
    }
    int out_position = index_to_position(out_index, out_strides, shape_size);

    // 3. Initialize the reduce_value to the output element
    for (int i = 0; i < shape_size; i++) {
      a_index[i] = out_index[i];
    }
    float out_i = reduce_value;

    // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
    // linear reduction of the values assigned to the thread
    for (a_index[reduce_dim] = threadIdx.x; a_index[reduce_dim] < a_shape[reduce_dim]; a_index[reduce_dim] += blockDim.x) {
      out_i = fn(fn_id, out_i, a_storage[index_to_position(a_index, a_strides, shape_size)]);
    }
    cache[threadIdx.x] = out_i;
    //binary reduction to combine threads in the block
    for (int offset = 1; offset < min(a_shape[reduce_dim], blockDim.x); offset <<= 1) {
      if (threadIdx.x % offset) return;

      if (threadIdx.x + offset >= a_shape[reduce_dim]) return; // these threads have no sibling

      // reduce with this thread's sibling
      out_i = fn(fn_id, out_i, cache[threadIdx.x + offset]);
      cache[threadIdx.x] = out_i;
      __syncthreads();
    }
    // 5. Write the reduced value to out memory
    out[out_position] = out_i;
}

__global__ void zipKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    int out_shape_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int a_shape_size,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
  /**
   * Zip function. Apply a binary function to elements of the input array a & b and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  out_shape_size: number of dimensions in the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  a_shape_size: number of dimensions in the input array
   *  b_storage: compact 1D array of size in_size
   *  b_shape: shape of the input array
   *  b_strides: strides of the input array
   *  b_shape_size: number of dimensions in the input array
   *  fn_id: id of the function to apply to each element of the a & b array
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

    // 1. Compute the position in the output array that this thread will write to
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= out_size) return;

    // 2. Convert the position to the out_index according to out_shape
    to_index(idx, out_shape, out_index, out_shape_size);
    for (int i = 0; i < out_shape_size; i++) {
        if (out_index[i] >= out_shape[i]) {
          return;
	    }
    }

    // 3. Calculate the position of element in out_array according to out_index and out_strides
    int out_position = index_to_position(out_index, out_strides, out_shape_size);

    // 4. Broadcast the out_index to the a_index according to a_shape
    broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
    for (int i = 0; i < a_shape_size; i++) {
        if (a_index[i] >= a_shape[i]) {
	          return;
	      }
    }

    // 5. Calculate the position of element in a_array according to a_index and a_strides
    int a_position = index_to_position(a_index, a_strides, a_shape_size);

    // 6. Broadcast the out_index to the b_index according to b_shape
    broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
    for (int i = 0; i < b_shape_size; i++) {
        if (b_index[i] >= b_shape[i]) {
	          return;
	      }
    }

    // 7.Calculate the position of element in b_array according to b_index and b_strides
    int b_position = index_to_position(b_index, b_strides, b_shape_size);

    // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
    out[out_position] = fn(fn_id, a_storage[a_position], b_storage[b_position]);
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* in_storage,
    int* in_shape,
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size,
      d_in, d_in_shape, d_in_strides,
      shape_size, fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    int out_shape_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = out_size;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size,
        d_a, d_a_shape, d_a_strides,
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}
